
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

//Host Code
__global__ void polynomial_expansion (float* poly, int degree, int n, float* array) { 
  
     int index = blockIdx.x * blockDim.x + threadIdx.x;
    if( index < n ){
        float polynomial = 0.0;
        float power = 1.0;
        for ( int i = 0; i < degree+1; ++i){
        polynomial += power * poly[i];
        power *= array[index];
        }
        array[index] = polynomial;
    }
}

//
//Referred Github Code
//
int main (int argc, char* argv[]) {
    ////TODO: add usage

    if (argc < 3) {
        std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
        return -1;
    }

    int n = atoi(argv[1]); 
    int degree = atoi(argv[2]);
    int nbiter = atoi(argv[3]);;

    float* array = new float[n];
    float* poly = new float[degree+1];
    for (int i=0; i<n; ++i) array[i] = 1.;

    for (int i=0; i<degree+1; ++i) poly[i] = 1.;

    float *dev_array, *dev_polynomial;
  
    std::chrono::time_point<std::chrono::system_clock> startTime, endTime;
    startTime = std::chrono::system_clock::now();
  	
	hipMallocManaged(&dev_array, n*sizeof(float));
  	hipMallocManaged(&dev_polynomial, (degree+1)*sizeof(float));
  
  	hipMemcpy(dev_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  	hipMemcpy(dev_polynomial, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);

  	polynomial_expansion<<<(n+255)/256, 256>>>(dev_polynomial, degree, n, dev_array);
  	hipMemcpy(array, dev_array, n*sizeof(float), hipMemcpyDeviceToHost);
       
    hipFree(dev_array);
    hipFree(dev_polynomial);

    hipDeviceSynchronize();

    endTime = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time = (endTime-startTime)/nbiter;	 
    
    {
        bool finish = true;
        for (int i=0; i< n; ++i) {
        if (fabs(array[i]-(degree+1))>0.01) {
            finish = false;
        }
        }
        if (!finish) std::cerr<<"Incorrect Result."<<std::endl;
    }

    std::cerr<<array[0]<<std::endl;
    std::cout<<n<<" "<<degree<<" "<<elapsed_time.count()<<std::endl;

    delete[] array;
    delete[] poly;

    return 0;
}